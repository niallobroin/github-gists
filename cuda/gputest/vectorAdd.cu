
#include <hip/hip_runtime.h>
#include <stdio.h>

/* Code given to Niall OByrnes by Zedong Wu from the KAUST SWAG
*/



// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
int main(int argc, char** argv)
{

    printf("Vector Addition\n");
    int N = 5000;
    size_t size = N * sizeof(float);
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    RandomInit(h_A, N);
    RandomInit(h_B, N);
    ( hipMalloc((void**)&d_A, size) );
    ( hipMalloc((void**)&d_B, size) );
    ( hipMalloc((void**)&d_C, size) );
    ( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    ( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    ( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    int i;
    double real=0.0;
    double error=0.0;
    for (i = 0; i < N; ++i) 
    {
	real+=pow(h_A[i]+h_B[i],2);
        error+=pow(h_A[i]+h_B[i]-h_C[i],2);
    }
    error=sqrt(error/real);
    if(error<1e-4)
    printf("passed with the error=%e\n",error);
    else
    printf("fail\n");
}
// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
